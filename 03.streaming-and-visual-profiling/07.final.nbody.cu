#include "hip/hip_runtime.h"
/*
 * This exercise sadly requires extra code (timer.h, files.h)
 * which I didn't have access. This code won't compile without them :(
 * Anyway, this is my optimized solution for the n-body problem:
 */
 
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

inline hipError_t checkCuda(hipError_t result) {
    if(result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        // break the program
        assert(result == hipSuccess);
    }
    return result;
}

/*
* Each body contains x, y, and z coordinate positions,
* as well as velocities in the x, y, and z directions.
*/

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
* Calculate the gravitational impact of all bodies in the system
* on all others.
*/

__global__ void bodyForce(Body *bodies, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (; i < n; i += stride) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = bodies[j].x - bodies[i].x;
            float dy = bodies[j].y - bodies[i].y;
            float dz = bodies[j].z - bodies[i].z;
            float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
        }

        bodies[i].vx += dt*Fx; 
        bodies[i].vy += dt*Fy; 
        bodies[i].vz += dt*Fz;
    }
}

__global__ void integratePos(Body *bodies, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (; i < n; i += stride) {
        bodies[i].x += bodies[i].vx*dt;
        bodies[i].y += bodies[i].vy*dt;
        bodies[i].z += bodies[i].vz*dt;
    }
}


int main(const int argc, const char** argv) {

    // The assessment will test against both 2<11 and 2<15.
    // Feel free to pass the command line argument 15 when you gernate ./nbody report files
    int nBodies = 2<<11;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    // The assessment will pass hidden initialized values to check for correctness.
    // You should not make changes to these files, or else the assessment will not work.
    const char * initialized_values;
    const char * solution_values;

    if (nBodies == 2<<11) {
        initialized_values = "files/initialized_4096";
        solution_values = "files/solution_4096";
    } else { // nBodies == 2<<15
        initialized_values = "files/initialized_65536";
        solution_values = "files/solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f; // Time step
    const int nIters = 10;  // Simulation iterations

    int bytes = nBodies * sizeof(Body);
    float *buf;

    // cuda initializations
    
    int deviceId;
    hipDeviceProp_t props;

    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&props, deviceId);
    
    dim3 blocks(props.multiProcessorCount * 32); // 80*32
    dim3 threads_per_block(props.warpSize * 8); // 256
    
    checkCuda(hipMallocManaged(&buf, bytes));
    
    // further assessment initializations
    
    Body *bodies = (Body*)buf;

    read_values_from_file(initialized_values, buf, bytes);

    double totalTime = 0.0;

    // prefetch to GPU
    checkCuda(hipMemPrefetchAsync(buf, bytes, deviceId));
    
    /*
    * This simulation will run for 10 cycles of time, calculating gravitational
    * interaction amongst bodies, and adjusting their positions to reflect.
    */

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        // synchronize from the last integration execution (to allow for cpu co-processing)
        checkCuda(hipDeviceSynchronize());
        
        /*
        * You will likely wish to refactor the work being done in `bodyForce`,
        * and potentially the work to integrate the positions.
        */

        bodyForce<<<blocks, threads_per_block>>>(bodies, dt, nBodies); // compute interbody forces
        checkCuda(hipGetLastError());
        checkCuda(hipDeviceSynchronize());

        /*
        * This position integration cannot occur until this round of `bodyForce` has completed.
        * Also, the next round of `bodyForce` cannot begin until the integration is complete.
        */

        integratePos<<<blocks, threads_per_block>>>(bodies, dt, nBodies); // integrate position
        checkCuda(hipGetLastError());

        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

    // prefetch back to CPU
    checkCuda(hipMemPrefetchAsync(buf, bytes, hipCpuDeviceId));
    
    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
    write_values_to_file(solution_values, buf, bytes);

    // You will likely enjoy watching this value grow as you accelerate the application,
    // but beware that a failure to correctly synchronize the device might result in
    // unrealistically high values.
    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    hipFree(buf);
}
 