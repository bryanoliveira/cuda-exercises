
#include <hip/hip_runtime.h>
#include <stdio.h>

inline void checkCuda(hipError_t result) {
    if(result != hipSuccess) printf("CUDA Error: %s\n", hipGetErrorString(result));
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for(; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  checkCuda(hipMallocManaged(&a, size));
  checkCuda(hipMallocManaged(&b, size));
  checkCuda(hipMallocManaged(&c, size));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threads_per_block = 256;
  size_t blocks = (N + threads_per_block - 1) / threads_per_block;
  addVectorsInto<<<blocks, threads_per_block>>>(c, a, b, N);
  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  checkCuda(hipFree(a));
  checkCuda(hipFree(b));
  checkCuda(hipFree(c));
}
