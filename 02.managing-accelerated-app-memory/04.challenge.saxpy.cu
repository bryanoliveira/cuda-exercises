#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define N 2048 * 2048 // Number of elements in each vector

inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

// Initialize memory
__global__ void initVectors(int * a, int * b, int * c) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for(; i < N; i += stride) {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }
}

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * EDIT: I made it run under 77 us :)
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (; i < N; i += stride)
        c[i] = 2 * a[i] + b[i];
}

int main()
{
    int *a, *b, *c;
    int size = N * sizeof (int); // The total number of bytes per vector
    
    int deviceId;
    hipDeviceProp_t props;
    
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&props, deviceId);

    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&c, size));

    int threads_per_block = props.warpSize * 8;
    int number_of_blocks = props.multiProcessorCount * 32; // using stride is better than arbitrary blocks
    
    // prefetch to gpu
    checkCuda(hipMemPrefetchAsync(a, size, deviceId));
    checkCuda(hipMemPrefetchAsync(b, size, deviceId));
    checkCuda(hipMemPrefetchAsync(c, size, deviceId));
    
    initVectors <<< number_of_blocks, threads_per_block >>>( a, b, c );
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());
    
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
